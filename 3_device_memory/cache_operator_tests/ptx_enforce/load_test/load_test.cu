#include <stdio.h>
#include <hip/hip_runtime.h>
#include <asm/unistd.h>
#include <fcntl.h>
#include <inttypes.h>
#include <linux/kernel-page-flags.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>
#include <sys/ioctl.h>
#include <sys/mount.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/sysinfo.h>
#include <sys/wait.h>
#include <time.h>
#include <unistd.h>
#include <vector>
#include <sys/time.h>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line,
		      bool abort = true)
{
    if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);
	if (abort)
	    exit(code);
    }
}


__global__ void kernel(int reads, size_t size,
		       volatile uint64_t * first, float *dsum)
{

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int threadId = blockId * (blockDim.x * blockDim.y)
	+ (threadIdx.y * blockDim.x) + threadIdx.x;

    int local_sum = 0;

    printf(" I am %d\n", threadId);

    while (reads--) {
	local_sum = first[0] + 666;
    }
	
   *dsum = local_sum;

}



int main(int argc, char **argv)
{

    int opt, BLOCKS = 1, THREADS = 1, reads = 1, size = 1;

    hipEvent_t start, stop;
    float elapsed_time;

    while ((opt = getopt(argc, argv, "b:t:r:s:")) != -1) {

		switch (opt) {

			case 'b':
			    BLOCKS = atoi(optarg);
			    break;
			case 't':
			    THREADS = atoi(optarg);
			    break;
			case 'r':
			    reads = atoi(optarg);
			    break;
			case 's':
			    size = atoi(optarg);
			    break;
			default:
			    fprintf(stderr, "Usage: %s -b [blocks] -t [threads] -r [reads] -s [size]\n",
				    argv[0]);
			    exit(EXIT_FAILURE);
		}
    }

    volatile uint64_t *h_mem;

    volatile uint64_t *d_mem;

    float *dsum;

    size_t h_mem_size = size * sizeof(uint64_t);

    gpuErrchk(hipEventCreate(&start));

    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipHostAlloc
	      ((void **) &h_mem, (size_t) h_mem_size, hipHostMallocMapped));

    gpuErrchk(hipMallocManaged((void **) &dsum, sizeof(float)));

    memset((void *)h_mem, 1, h_mem_size);


    gpuErrchk(hipHostGetDevicePointer
	      ((void **) &d_mem, (void *) h_mem, 0));

    *dsum = 0;

    gpuErrchk(hipEventRecord(start, 0));

    kernel <<< BLOCKS, THREADS >>> (reads, size, d_mem, dsum);

    gpuErrchk(hipDeviceSynchronize());

    assert(*dsum != 0);

    gpuErrchk(hipEventRecord(stop, 0));

    gpuErrchk(hipEventSynchronize(stop));

    gpuErrchk(hipEventElapsedTime(&elapsed_time, start, stop));

    printf("\n\tElapsed time : %f ms \n\n", elapsed_time);

    return 0;
}
