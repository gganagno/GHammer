#include <asm/unistd.h>
#include <assert.h>
#include <errno.h>
#include <fcntl.h>
#include <inttypes.h>
#include <linux/kernel-page-flags.h>
#include <map>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>
#include <sys/ioctl.h>
#include <sys/mount.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/sysinfo.h>
#include <sys/wait.h>
#include <time.h>
#include <unistd.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <assert.h>
#include <pthread.h>


#define MB(x) ((x) << 20)
#define GB(x) ((x) << 30)

#define B2MB(x) (x/1024/1024)
#define MS2SEC(x) (x/1000)
#define MB2B(x) (x*1024*1024)



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line,
		      bool abort = true)
{
    if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);
	if (abort)
	    exit(code);
    }
}






__global__ void kernel(volatile char *dev_mem,volatile int * d_mapping, uint64_t l2_size,
		       uint64_t * d_sum, int number_of_threads)
{

    int i;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int threadId = blockId * (blockDim.x * blockDim.y)
	+ (threadIdx.y * blockDim.x) + threadIdx.x;


    int index = threadId * l2_size / number_of_threads;

    printf("\t\t<<thread %d>> reads [%d-%d] \n", threadId, index,
	   index + (l2_size / number_of_threads));


    for (i = 0; i < l2_size / number_of_threads; ++i) {

	   d_mapping[0] += dev_mem[index + i];
    }

    *d_sum = d_mapping[0];
}


int main(int argc, char **argv)
{



    int opt, BLOCKS = 1, THREADS = 1;
    long l2_size = 0;
    hipEvent_t start, stop;
    float elapsed_time;
    volatile char *device_mem ;


    while ((opt = getopt(argc, argv, "b:t:l:f:")) != -1) {

	switch (opt) {

	case 'b':
	    BLOCKS = atoi(optarg);
	    break;
	case 't':
	    THREADS = atoi(optarg);
	    break;
	case 'l':
	    l2_size = MB2B(atoi(optarg));
	    break;
	case 'f':
	    l2_size = atoi(optarg);
	    break;


	default:
	    fprintf(stderr,
		    "Usage: %s -b [blocks] -t [threads] -d [debug] -f [l2_size] \n",
		    argv[0]);
	    exit(EXIT_FAILURE);
	}
    }


    uint64_t *d_sum;

    gpuErrchk(hipMallocManaged((void **) &d_sum, sizeof(uint64_t)));


    gpuErrchk(hipEventCreate(&start));

    gpuErrchk(hipEventCreate(&stop));


    volatile int * h_mapping;

    gpuErrchk(hipHostAlloc( (void**)&h_mapping, sizeof(int), hipHostMallocMapped));

    volatile int * d_mapping;

    gpuErrchk(hipHostGetDevicePointer((void**)&d_mapping,(void*)h_mapping,0));

    printf("\n\n\thipMalloc device_mem : %zu bytes\n\n", l2_size);

    gpuErrchk(hipMalloc((void **) &device_mem, l2_size));

    gpuErrchk(hipMemset((void *) device_mem, 1, l2_size));



    *d_sum = 0;

    gpuErrchk(hipEventRecord(start, 0));




    kernel <<< BLOCKS, THREADS >>> (device_mem,d_mapping,l2_size,d_sum,
		BLOCKS * THREADS);





    gpuErrchk(hipDeviceSynchronize());


    gpuErrchk(hipEventRecord(stop, 0));

    gpuErrchk(hipEventSynchronize(stop));


    assert(*d_sum != 0);


    gpuErrchk(hipEventElapsedTime(&elapsed_time, start, stop));

    printf("\n\tElapsed time : %f ms \n\n", elapsed_time);

    gpuErrchk(hipEventDestroy(start));

    gpuErrchk(hipEventDestroy(stop));

    hipFree((void *) device_mem);
    hipFree((void *) d_sum);
}
