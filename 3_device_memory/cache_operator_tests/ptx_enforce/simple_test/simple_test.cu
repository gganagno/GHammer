#include <stdio.h>
#include <hip/hip_runtime.h>
#include <asm/unistd.h>
#include <fcntl.h>
#include <inttypes.h>
#include <linux/kernel-page-flags.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>
#include <sys/ioctl.h>
#include <sys/mount.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/sysinfo.h>
#include <sys/wait.h>
#include <time.h>
#include <unistd.h>
#include <vector>
#include <sys/time.h>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line,
		      bool abort = true)
{
    if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);
	if (abort)
	    exit(code);
    }
}


__global__ void kernel(uint64_t * d_mem, float *dsum)
{

    *d_mem = 666;
    *dsum = *d_mem;

}



int main(int argc, char **argv)
{


    uint64_t *h_mem;
    uint64_t *d_mem;

    float * dsum;
	int BLOCKS = 1 , THREADS =1;

    gpuErrchk(hipHostAlloc
	      ((void **) &h_mem, sizeof(uint64_t), hipHostMallocMapped));

    gpuErrchk(hipMallocManaged((void **) &dsum, sizeof(float)));


    gpuErrchk(hipHostGetDevicePointer
	      ((void **) &d_mem, (void *) h_mem, 0));


    *dsum = 0;

    *h_mem = 555;
    printf("h_mem before kernel: %u\n",*h_mem );

    kernel <<< BLOCKS, THREADS >>> (d_mem, dsum);

    
    gpuErrchk(hipDeviceSynchronize());

    printf("h_mem after kernel: %u\n",*h_mem );
    
    assert(*dsum != 0);

    return 0;
}
