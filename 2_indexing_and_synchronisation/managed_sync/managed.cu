#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <vector>
#include <assert.h>
#include <signal.h>


        
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line,
              bool abort = true)
{
    if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
        file, line);
    if (abort)
        exit(code);
    }
}




__global__ void kernel(int number_of_threads,int * managed)
{
   
    int index = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
     + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    printf("[D] I am %d\n",index );

    *managed = 1;

}





int main(int argc, char **argv)
{

    int opt, BLOCKS = 1, THREADS = 1, error = 0;


    while ((opt = getopt(argc, argv, "b:t:e:")) != -1) {

        switch (opt) 
        {
        case 'b':
            BLOCKS = atoi(optarg);
            break;
        case 't':
            THREADS = atoi(optarg);
            break;
        case 'e':
            error = atoi(optarg);
            break;

        default:
            fprintf(stderr, "Usage: %s -b [blocks] -t [threads]\n",
                argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    int * managed;

    gpuErrchk(hipMallocManaged((void **) &managed,sizeof(int)));

    *managed = 0;

    kernel <<< BLOCKS, THREADS >>> (BLOCKS * THREADS, managed);

    if(error){

        *managed = 2;
        gpuErrchk(hipDeviceSynchronize());

    }else{
        printf("[H] before cudaDeviceSynchronize\n");
        gpuErrchk(hipDeviceSynchronize());
        assert(*managed != 0);
        printf("[H] After hipDeviceSynchronize managed:%d\n",*managed);
        *managed = 2;
        printf("[H] After cpu access managed:%d\n",*managed);
        
    }


    return 0;
 
}
