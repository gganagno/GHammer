#include "hip/hip_runtime.h"

#include "included.h"
#include "thread_indexing.h"

pthread_t posix_thread;

volatile uint64_t * glb_ptr [4];

int fence=0;

void * thread_func(void * arg){

  pthread_t self_id;
  self_id=pthread_self();

  printf("\tIm the POSIXthread %u\n",self_id);

  while(1){

    if(glb_ptr[0] != NULL && glb_ptr[1] != NULL && glb_ptr[2] != NULL && glb_ptr[3] != NULL){

      //printf("[T] glb_ptr[0]: %p\nglb_ptr[1]: %p\nglb_ptr[2]: %p\nglb_ptr[3]: %p\n",glb_ptr[0],glb_ptr[1],glb_ptr[2],glb_ptr[3]);
       asm volatile(
            "clflush (%0);\n\t"
          "clflush (%1);\n\t"
          "clflush (%2);\n\t"
          "clflush (%3);\n\t"
            : : "r" (glb_ptr[0]), "r" (glb_ptr[1]), "r" (glb_ptr[2]), "r" (glb_ptr[3]) : "memory");
    }
  }
}


void SetupMapping(uint64_t * mapping_size,volatile void** mapping) {

  *mapping_size = GetPhysicalMemorySize() * fraction_of_physical_memory;


  if(!memory_type_flag){

    printf("[!] Allocating with hipHostAlloc %lu MB\n",B2MB(*mapping_size));

    gpuErrchk(hipHostAlloc(mapping,(size_t)*mapping_size,hipHostMallocMapped));


  }else{

    printf("[!] Allocating with hipHostMalloc %lu MB\n",B2MB(*mapping_size));

    gpuErrchk(hipHostMalloc(mapping,*mapping_size));


  }

  for (uint64_t index = 0; index < *mapping_size; index += 0x1000) {

    uint64_t* temporary = (uint64_t *)( (uint8_t *)(*mapping)  + index);

    temporary[0] = index;
  }

}




__global__ void hammer(

  volatile uint64_t * first1,
  volatile uint64_t * first2,
  volatile uint64_t * second1,
  volatile uint64_t * second2,
  uint64_t * sum,
  uint64_t number_of_reads,
  int fence
  ) {


    uint64_t local_sum = 0;

    int thId = getGlobalIdx_2D_2D();

    //printf("I am GPUthread %d \n",thId);

    *sum =0;

     while (number_of_reads-- > 0) 
    {
      local_sum += first1[thId%64];
      local_sum += second1[thId%64];

      if(fence){
        __threadfence_system();
      }

    }

   *sum = local_sum + 1;
}




void HammerAllReachablePages(

  uint64_t presumed_row_size, 
  volatile void * memory_mapping, 
  uint64_t memory_mapping_size, 
  uint64_t number_of_reads) 

{

  CpuTimer table_time;

  std::vector<std::vector<uint8_t*> > pages_per_row;

  uint64_t total_bitflips = 0;

  int i=0,j=0,z=0;

  double time[64];

  double page_time=0, average =0;

  uint64_t * d_sum;

  uint64_t * h_first[2], * h_second[2];

  volatile uint64_t * d_first1 = NULL ;
  volatile uint64_t * d_first2 = NULL ;
  volatile uint64_t * d_second1 = NULL;
  volatile uint64_t * d_second2 = NULL;

 

  pages_per_row.resize(memory_mapping_size / presumed_row_size);

  int pagemap = open("/proc/self/pagemap", O_RDONLY);

  assert(pagemap >= 0);


  //filling the pointers table
    for (uint64_t offset = 0; offset < memory_mapping_size; offset += 0x1000) {

    uint8_t* virtual_address = (uint8_t*)(memory_mapping) + offset;

    uint64_t page_frame_number = GetPageFrameNumber(pagemap, virtual_address);

    uint64_t physical_address = page_frame_number * 0x1000;

    uint64_t presumed_row_index = physical_address / presumed_row_size;
    

    if (presumed_row_index > pages_per_row.size()) {
      pages_per_row.resize(presumed_row_index);
    }
  
        pages_per_row[presumed_row_index-1].push_back(virtual_address);      
    }
    //done finding the pointers

    printf("[!] pointers for hammering took %f ms\n",table_time.get_diff_ms());
  
    //device memory for writing the read data

  gpuErrchk(hipMallocManaged((void**)&d_sum, sizeof(uint64_t)));


  pthread_create(&posix_thread,NULL,&thread_func,NULL);


  for (uint64_t row_index = 0; row_index + 2 < pages_per_row.size(); ++row_index) {

    CpuTimer row_time;


      if ((pages_per_row[row_index].size() != 64) || (pages_per_row[row_index+2].size() != 64)) {

          continue;

      }else if (pages_per_row[row_index+1].size() == 0) {

          printf("[!] Can't hammer row %ld,got no pages from that row\n",row_index+1);
          continue;
      }

    printf("[!] Hammering rows %ld/%ld/%ld of %ld (got %ld/%ld/%ld pages)\n", 
      row_index, row_index+1, row_index+2, pages_per_row.size(), 
      pages_per_row[row_index].size(), pages_per_row[row_index+1].size(), 
      pages_per_row[row_index+2].size());



      for (uint8_t * first_row_page : pages_per_row[row_index]) {

        CpuTimer page_with_row_time;
        j=0;
             // Iterate over all pages we have for the second row.
          for (uint8_t* second_row_page : pages_per_row[row_index+2]) {


               // Set all the target pages to 0xFF.
            for (uint8_t* target_page : pages_per_row[row_index+1]) {
                 memset(target_page, 0xFF, 0x1000);
            }

          

            glb_ptr[0] = h_first[0] = (uint64_t *) first_row_page;
            glb_ptr[1] = h_first[1] = (uint64_t *) first_row_page + 0x1000;

            glb_ptr[2] = h_second[0] = (uint64_t *) second_row_page;
            glb_ptr[3] = h_second[1] = (uint64_t *) second_row_page + 0x1000;

            gpuErrchk(hipHostGetDevicePointer((void**)&d_first1,(void*)h_first[0],0));
            gpuErrchk(hipHostGetDevicePointer((void**)&d_first2,(void*)h_first[1],0));
            gpuErrchk(hipHostGetDevicePointer((void**)&d_second1,(void*)h_second[0],0));
            gpuErrchk(hipHostGetDevicePointer((void**)&d_second2,(void*)h_second[1],0));


            GpuTimer g_timer;

            g_timer.Start();

            *d_sum=0;

            //printf("1dsum is %u \n", *d_sum);

            hammer<<<BLOCKS,THREADS>>>(d_first1,d_first2,d_second1,d_second2,d_sum,number_of_reads,fence);
        
            gpuErrchk(hipDeviceSynchronize());

            g_timer.Stop();

            //printf("2dsum is %u \n", *d_sum);

          assert(* d_sum != 0);

          time[i] = g_timer.Elapsed();

          if(hammering_time)
            printf("[H] hammering : %f ms\n\n",time[i]);
  
          ++i;

            uint64_t number_of_bitflips_in_target = 0;
          
          for (const uint8_t* target_page : pages_per_row[row_index+1]) {

              for (uint32_t index = 0; index < 0x1000; ++index) {
                  
                  if (target_page[index] != 0xFF){
                      ++number_of_bitflips_in_target;
                    }
              }
          }

          if (number_of_bitflips_in_target) {

          printf("[!] Found %ld flips in row %ld (%lx to %lx) when hammering "
          "%lx and %lx\n", number_of_bitflips_in_target, row_index+1,
          ((row_index+1)*presumed_row_size), 
          ((row_index+2)*presumed_row_size)-1,
          GetPageFrameNumber(pagemap, first_row_page)*0x1000, 
          GetPageFrameNumber(pagemap, second_row_page)*0x1000);

          total_bitflips += number_of_bitflips_in_target;
        } // if

      } // second row for

      page_time = page_with_row_time.get_diff_ms();


      for(z = 0 ; z < pages_per_row[row_index+2].size() ; z++){
        average += time[z];
      }

      average /= 64 ; 
  

      if(debug){

            printf("\t %p with row %lu took %f secs \n",first_row_page, row_index +2 ,MS2SEC(page_time));
          
            printf("%d:[!] hammering a set of pointers (median): %f ms\n",++j,average);
       
      }
        
        i = average = page_time = 0;
      } // first_row for

      if(debug)
          printf("\t hammering row %lu took %f ms \n",row_index+1,MS2SEC(row_time.get_diff_ms()));
  } // end of row_index for

}//end



void HammerAllReachableRows(uint64_t number_of_reads) {
  

  uint64_t mapping_size;

  volatile void * mapping;

  SetupMapping(&mapping_size, &mapping);

  HammerAllReachablePages(1024*256, mapping, mapping_size,number_of_reads);

}



void prepare(){

  setvbuf(stdout, NULL, _IONBF, 0);

  signal(SIGALRM, HammeredEnough);

  alarm(number_of_seconds_to_hammer);

}



void cuda_prepare(){

  hipDeviceProp_t prop;

  hipGetDeviceProperties(&prop, 0);

  if (!prop.canMapHostMemory) exit(0);

  printf("[!] Compute : %d.%d \n", prop.major, prop.minor);

  hipSetDevice(0);

  hipSetDeviceFlags(hipDeviceMapHost);

}






int main(int argc, char** argv) {

  int opt;

  while ((opt = getopt(argc, argv, "b:t:d:m:n:f:h:")) != -1) {

    switch (opt) {

      case 'b':
        BLOCKS = atoi(optarg);
        break;
      case 't':
        THREADS = atoi(optarg);
        break;
      case 'd':
        debug =atoi(optarg);
        break;
      case 'm':
        memory_type_flag = atoi(optarg);
        break;
      case 'n':
        number_of_reads =atoi(optarg);
        break;
      case 'h':
        hammering_time = atoi(optarg);
        break;
      case 'f':
        fence  = atoi(optarg);
        break;
      default:
        fprintf(stderr, "Usage: %s -b [blocks] -t [threads] -d [debug] [-m memory_type] [-h hammering_time] \n",argv[0]);
        exit(EXIT_FAILURE);
    }
  }


   uint64_t uid = getuid();

  if(uid != 0){
    fprintf(stderr,"Must be in root\n");
    exit(EXIT_FAILURE);
  }

  printf("\n[!] BLOCKS: %d THREADS PER BLOCK : %d \n",BLOCKS,THREADS);

  if(debug){
    printf("\n[!] DEBUG_MODE\n");
  }else{
    printf("\n[!] RUN_MODE\n");
  }

  printf("[!] fraction : %.1f\n", fraction_of_physical_memory);
  printf("[!] reads : %lu\n", number_of_reads);

    prepare();
    cuda_prepare();

    HammerAllReachableRows(number_of_reads);

}

