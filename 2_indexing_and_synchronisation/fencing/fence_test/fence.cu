#include <stdio.h>
#include <hip/hip_runtime.h>
#include <asm/unistd.h>
#include <fcntl.h>
#include <inttypes.h>
#include <linux/kernel-page-flags.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>
#include <sys/ioctl.h>
#include <sys/mount.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/sysinfo.h>
#include <sys/wait.h>
#include <time.h>
#include <unistd.h>
#include <vector>
#include <sys/time.h>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line,
		      bool abort = true)
{
    if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);
	if (abort)
	    exit(code);
    }
}


__global__ void kernel(int number_of_threads, float * dsum ,volatile int * d_mapping, int cnt, int fence_system_flag, int fence_block_flag)
{
    int i;

    /*printf("D: i am [%d] \n", blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
     + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);
    */

    for(i=0; i < cnt ; i++){
        *dsum = i;
        d_mapping[0] = *dsum;

        if(fence_system_flag){
            __threadfence_system();
        }
        if(fence_block_flag){
            __threadfence_block();
        }
    }
}


int main(int argc, char **argv)
{

    int opt, BLOCKS = 1, THREADS = 1, cnt =10000, fence_system_flag =0, fence_block_flag = 0;

    hipEvent_t start, stop;
    float elapsed_time =0;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));


    while ((opt = getopt(argc, argv, "b:t:n:f:s:")) != -1) {

    	switch (opt) {

    	case 'b':
    	    BLOCKS = atoi(optarg);
    	    break;
    	case 't':
    	    THREADS = atoi(optarg);
    	    break;
        case 'n':
            cnt = atoi(optarg);
            break;
        case 'f':
            fence_system_flag = atoi(optarg);
            break;
        case 's':
            fence_block_flag = atoi(optarg);
            break;

    	default:
    	    fprintf(stderr, "Usage: %s -b [blocks] -t [threads] -n [count of iterations] -f [fence_system] -s [fence_block]\n",
    		    argv[0]);
    	    exit(EXIT_FAILURE);
    	}
    }

    float * dsum;

    gpuErrchk(hipMallocManaged((void **) &dsum, sizeof(uint64_t)));


    volatile int * h_mapping;

    gpuErrchk(hipHostAlloc( (void**)&h_mapping, sizeof(volatile int), hipHostMallocMapped));

    volatile int * d_mapping;

    gpuErrchk(hipHostGetDevicePointer((void**)&d_mapping,(void*)h_mapping,0));


    *dsum = 0;

    hipEventRecord(start, 0);

    kernel <<< BLOCKS, THREADS >>> (BLOCKS * THREADS,dsum,d_mapping, cnt, fence_system_flag, fence_block_flag);


    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipEventRecord(stop, 0));

    gpuErrchk(hipEventSynchronize(stop));

    gpuErrchk(hipEventElapsedTime
              (&elapsed_time, start, stop));


    assert(*dsum != 0);

    printf("H: elapsed_time is : %f \n", elapsed_time);

    hipEventDestroy(start);
    
    hipEventDestroy(stop);

    return 0;
}
